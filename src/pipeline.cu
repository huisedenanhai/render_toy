#include "hip/hip_runtime.h"
#include "pipeline.h"
#include "random.h"
#include "vec_math.h"
#include <optix.h>

using namespace dev;

extern "C" {
__constant__ LaunchParams g_LaunchParams;
}

__device__ __forceinline__ float2 rect_lerp(const Rect &rect,
                                            float u,
                                            float v) {
  return make_float2(rect.x + rect.width * u, rect.y + rect.height * v);
}

__device__ __forceinline__ uint2 current_pixel() {
  auto launchIndex = optixGetLaunchIndex();
  const auto &frame = g_LaunchParams.outputFrame;
  auto pixel =
      make_uint2(launchIndex.x + frame.tile.x, launchIndex.y + frame.tile.y);
  return pixel;
}

__device__ __forceinline__ unsigned int pixel_index(const uint2 &pixel) {
  return pixel.y * g_LaunchParams.outputFrame.width + pixel.x;
}

__device__ __forceinline__ float3 &pixel_value(const uint2 &pixel) {
  auto index = pixel_index(pixel);
  return g_LaunchParams.outputFrame.buffer[index];
}

__device__ __forceinline__ float3 &current_pixel_value() {
  return pixel_value(current_pixel());
}

__device__ __forceinline__ float
sample_camera_ray(unsigned int &randState, float3 &origin, float3 &direction) {
  const auto &cam = g_LaunchParams.camera;
  const auto &frame = g_LaunchParams.outputFrame;
  auto width = (float)frame.width;
  auto height = (float)frame.height;
  auto pixelIndex = current_pixel();
  auto pixel = make_float2(pixelIndex.x, pixelIndex.y);
  // jitter pixel position
  pixel.x += rnd(randState);
  pixel.y += rnd(randState);

  auto canvasXY = rect_lerp(cam.canvas, pixel.x / width, pixel.y / height);
  auto dir = cam.right * canvasXY.x + cam.up * canvasXY.y - cam.back;

  origin = cam.position;
  direction = normalize(dir);
  return 1.0f; // w / pdf
}

struct TangentSpace {
  float3 origin;
  float3 dpdu;
  float3 dpdv;
  float3 n;
};

struct Ray {
  float3 origin;
  float min;
  float3 direction;
  float max;
};

struct RayPayload {
  bool finish;
  Ray ray;
  int length;
  float3 weight;
  float3 color;
  unsigned int seed;
};

__device__ __forceinline__ void
unpack_ptr(void *ptr, unsigned int &p0, unsigned int &p1) {
  p0 = ((size_t)ptr) >> 32;
  p1 = ((size_t)ptr) & (unsigned int)(-1);
}

__device__ __forceinline__ void *pack_ptr(unsigned int p0, unsigned int p1) {
  size_t p = (((size_t)p0) << 32) | (size_t)p1;
  return (void *)p;
}

__device__ __forceinline__ RayPayload *get_prd() {
  return (RayPayload *)pack_ptr(optixGetPayload_0(), optixGetPayload_1());
}

// z goes upward
__device__ __forceinline__ float
uniform_sample_hemisphere(unsigned int &randState, float3 &d) {
  auto theta = 2.0f * Pi * rnd(randState);
  auto z = rnd(randState);
  auto r = sqrtf(max(0.0f, 1.0f - z * z));
  d.x = r * cosf(theta);
  d.y = r * sinf(theta);
  d.z = z;
  return 0.5f * InvPi;
}

// z goes upward
__device__ __forceinline__ float
cosine_sample_hemisphere(unsigned int &randState, float3 &d) {
  auto theta = 2.0f * Pi * rnd(randState);
  auto r = sqrtf(rnd(randState));
  auto z = sqrtf(1 - r * r);
  d.x = r * cosf(theta);
  d.y = r * sinf(theta);
  d.z = z;
  return z * InvPi;
}

extern "C" __device__ void __raygen__entry() {
  auto pixelIndex = current_pixel();
  RayPayload prd{};
  prd.seed = tea<4>(pixel_index(pixelIndex), 114514);

  unsigned int p0, p1;
  unpack_ptr(&prd, p0, p1);

  float3 pixelColor = make_float3(0.0f, 0.0f, 0.0f);
  auto spp = g_LaunchParams.spp;

  for (int i = 0; i < spp; i++) {
    prd.finish = false;
    prd.length = 0;
    prd.weight = make_float3(1.0f, 1.0f, 1.0f);
    prd.color = make_float3(0.0f, 0.0f, 0.0f);

    auto w = sample_camera_ray(prd.seed, prd.ray.origin, prd.ray.direction);
    const auto &scene = g_LaunchParams.scene;
    prd.ray.min = scene.epsilon;
    prd.ray.max = scene.extent;

    while (!prd.finish) {
      prd.length++;
      optixTrace(scene.gas,
                 prd.ray.origin,
                 prd.ray.direction,
                 prd.ray.min, // tmin
                 prd.ray.max, // tmax
                 0,           // ray time
                 255,         // mask
                 OPTIX_RAY_FLAG_NONE,
                 0, // sbt offset
                 1, // sbt stride
                 0, // miss index
                 p0,
                 p1);
    }
    pixelColor += w * prd.color;
  }
  current_pixel_value() = pixelColor / (float)spp;
}

extern "C" __device__ void __miss__entry() {
  auto data = (MissData *)optixGetSbtDataPointer();
  auto prd = get_prd();

  prd->color += prd->weight * data->color;
  prd->finish = true;
}

extern "C" __device__ void __exception__entry() {
  auto data = (ExceptionData *)optixGetSbtDataPointer();
  current_pixel_value() = data->errorColor;
}

extern "C" __device__ void __closesthit__entry() {
  auto prd = get_prd();

  // init geom
  auto data = (HitGroupData *)optixGetSbtDataPointer();
  auto primId = optixGetPrimitiveIndex();
  auto gas = optixGetGASTraversableHandle();
  auto uv = optixGetTriangleBarycentrics();
  float3 v[3];
  optixGetTriangleVertexData(gas, primId, 0, 0, v);

  // calculate tangent space
  TangentSpace ts;
  ts.origin = v[0] * (1 - uv.x - uv.y) + v[1] * uv.x + v[2] * uv.y;
  ts.dpdu = v[1] - v[0];
  ts.dpdv = v[2] - v[0];
  ts.n = normalize(cross(ts.dpdu, ts.dpdv));

  prd->color += prd->weight * data->emission;
  // Russian Roulette
  // don't make the prob too small
  auto continueRate = min(max(length(prd->weight), 0.03f), 1.0f);
  bool rrFinish = prd->length >= 3 && rnd(prd->seed) > continueRate;
  prd->finish = prd->length >= g_LaunchParams.maxPathLength || rrFinish;
  prd->weight /= continueRate;
  // calculate next ray
  float3 d;
  auto pdf = cosine_sample_hemisphere(prd->seed, d);
  float3 localX = normalize(ts.dpdu);
  float3 localZ = faceforward(ts.n, -prd->ray.direction, ts.n);
  float3 localY = normalize(cross(ts.n, localX));
  auto nextDir = normalize(localX * d.x + localY * d.y + localZ * d.z);
  // init next ray
  auto &scene = g_LaunchParams.scene;
  prd->ray.origin = ts.origin + localZ * scene.epsilon;
  prd->ray.direction = nextDir;
  prd->ray.min = scene.epsilon;
  prd->ray.max = scene.extent;
  // attenuate factor, some terms are cancelled out as
  // hemisphere are cosine sampled
  prd->weight *= data->baseColor;
}

extern "C" __device__ void __anyhit__entry() {}